
#include <hip/hip_runtime.h>


__global__ void sinWaveVboKernel(float4 *pos, unsigned int width, unsigned int height, float animTime){
	
	unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

	float freq = 4.0;
	float u = x / float(width);
	float v = y / float(height);
	
	u = u * 2.0 - 1.0;
	v = v * 2.0 - 1.0;

	float w = sinf(freq * u + animTime) * cosf(freq * v + animTime) * 0.5 ;

	
	pos[y * width + x] = make_float4(u, w, v, 1.0);

}



void launchCudaKernel(float4* pos, unsigned int width, unsigned int height, float time){

	dim3 block(8, 8, 1);
	dim3 grid(width / block.x, height / block.y, 1);

	sinWaveVboKernel<<<grid, block>>>(pos, width, height, time);


}
